#include "hip/hip_runtime.h"
#include "solveCudaSharedMemory.cuh"

#include <stdio.h>
#include <chrono>

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                                 SOLVER                                                           //
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Solve::cudaSharedMemory(int *res, const int *matrix, const int sideSize)
{
	hipError_t cudaStatus = Solve::Internal::cudaSharedMemory(res, matrix, sideSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Solve::cuda failed!\n");
		fflush(stdout);
		return;
	}
}

void Solve::testCudaSharedMemory(int* res, const int* matrix, const int sideSize)
{
    hipError_t cudaStatus = Solve::Internal::testCudaSharedMemory(res, matrix, sideSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Solve::cuda failed!\n");
        fflush(stdout);
        return;
    }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                               INTERNAL                                                           //
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void Solve::Internal::computeShared(int* res, const int* arr, const int size)
{
    extern __shared__ int block[];
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < size) {
        block[threadIdx.x] = 0;
        for (int row = 0; row < size; ++row) {
            const int elemIdx = col * size + row;
            if (arr[elemIdx] == 0)
                ++block[col];
        }
    }
    res[col] = block[threadIdx.x];
}


///////////////////////////////
//       BASE FUNCTION       //
///////////////////////////////
hipError_t Solve::Internal::cudaSharedMemory(int* res, const int* arr, const int size)
{
    int* dev_arr = 0;
    int* dev_res = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }

    
    // Allocate GPU buffers for 2 arrays (one input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_res, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_arr, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }
   
    
    // Copy input array from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_arr, arr, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }
    

    const int block_size = deviceProp.maxThreadsPerBlock;
    const int num_blocks = size / block_size + 1;
    int resBlockSize = block_size;
    if (num_blocks == 1)
        resBlockSize = size;

    
    // Launch a kernel on the GPU with one thread for each column.
    computeShared <<<num_blocks, resBlockSize, resBlockSize * sizeof(int)>>> (dev_res, dev_arr, resBlockSize);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    
    // Copy output array from GPU buffer to host memory.
    cudaStatus = hipMemcpy(res, dev_res, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

Error:
    hipFree(dev_res);
    hipFree(dev_arr);

    return cudaStatus;
}

///////////////////////////////
//       TEST FUNCTION       //
///////////////////////////////
hipError_t Solve::Internal::testCudaSharedMemory(int* res, const int* arr, const int size)
{
    int* dev_arr = 0;
    int* dev_res = 0;
    hipError_t cudaStatus;

    hipEvent_t eAllocStart, eAllocStop;
    hipEvent_t eCopyStart, eCopyStop;
    hipEvent_t eComputeStart, eComputeStop;
    hipEvent_t eReciveStart, eReciveStop;
    hipEvent_t eFreeStart, eFreeStop;

    hipEventCreate(&eAllocStart);
    hipEventCreate(&eAllocStop);

    hipEventCreate(&eCopyStart);
    hipEventCreate(&eCopyStop);

    hipEventCreate(&eComputeStart);
    hipEventCreate(&eComputeStop);

    hipEventCreate(&eReciveStart);
    hipEventCreate(&eReciveStop);

    hipEventCreate(&eFreeStart);
    hipEventCreate(&eFreeStop);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }


    const std::chrono::system_clock::time_point startTimeAlloc = std::chrono::system_clock::now();
    // Allocate GPU buffers for two arrays (one input, one output)    .
    hipEventRecord(eAllocStart);
    cudaStatus = hipMalloc((void**)&dev_res, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_arr, size * size * sizeof(int));
    hipEventRecord(eAllocStop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }
    const std::chrono::system_clock::time_point endTimeAlloc = std::chrono::system_clock::now();


    const std::chrono::system_clock::time_point startTimeCopy = std::chrono::system_clock::now();
    // Copy input arrays from host memory to GPU buffers.
    hipEventRecord(eCopyStart);
    cudaStatus = hipMemcpy(dev_arr, arr, size * size * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(eCopyStop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }
    const std::chrono::system_clock::time_point endTimeCopy = std::chrono::system_clock::now();


    const int block_size = deviceProp.maxThreadsPerBlock;
    const int num_blocks = size / block_size + 1;
    int resBlockSize = block_size;
    if (num_blocks == 1)
        resBlockSize = size;


    const std::chrono::system_clock::time_point startTimeCompute = std::chrono::system_clock::now();
    // Launch a kernel on the GPU with one thread for each column.
    hipEventRecord(eComputeStart);
    computeShared <<<num_blocks, resBlockSize, resBlockSize * sizeof(int)>>> (dev_res, dev_arr, resBlockSize);
    hipEventRecord(eComputeStop);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    const std::chrono::system_clock::time_point endTimeCompute = std::chrono::system_clock::now();


    const std::chrono::system_clock::time_point startTimeRecive = std::chrono::system_clock::now();
    // Copy output array from GPU buffer to host memory.
    hipEventRecord(eReciveStart);
    cudaStatus = hipMemcpy(res, dev_res, size * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(eReciveStop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }
    const std::chrono::system_clock::time_point endTimeRecive = std::chrono::system_clock::now();


Error:
    const std::chrono::system_clock::time_point startTimeFree = std::chrono::system_clock::now();
    hipEventRecord(eFreeStart);
    hipFree(dev_res);
    hipFree(dev_arr);
    hipEventRecord(eFreeStop);
    const std::chrono::system_clock::time_point endTimeFree = std::chrono::system_clock::now();


    float eAllocTime;
    float eCopyTime;
    float eComputeTime;
    float eReciveTime;
    float eFreeTime;

    hipEventSynchronize(eAllocStop);
    hipEventSynchronize(eCopyStop);
    hipEventSynchronize(eComputeStop);
    hipEventSynchronize(eReciveStop);
    hipEventSynchronize(eFreeStop);
    hipEventElapsedTime(&eAllocTime, eAllocStart, eAllocStop);
    hipEventElapsedTime(&eCopyTime, eCopyStart, eCopyStop);
    hipEventElapsedTime(&eComputeTime, eComputeStart, eComputeStop);
    hipEventElapsedTime(&eReciveTime, eReciveStart, eReciveStop);
    hipEventElapsedTime(&eFreeTime, eFreeStart, eFreeStop);

    printf("Alloc time CUDA:   %d s\n                   %d ms\n                   %f ms (CUDA events)\n                   %d ns\n",
        std::chrono::duration_cast<std::chrono::seconds>(endTimeAlloc - startTimeAlloc).count(),
        std::chrono::duration_cast<std::chrono::milliseconds>(endTimeAlloc - startTimeAlloc).count(),
        eAllocTime,
        std::chrono::duration_cast<std::chrono::nanoseconds>(endTimeAlloc - startTimeAlloc).count());
    printf("Copy time CUDA:    %d s\n                   %d ms\n                   %f ms (CUDA events)\n                   %d ns\n",
        std::chrono::duration_cast<std::chrono::seconds>(endTimeCopy - startTimeCopy).count(),
        std::chrono::duration_cast<std::chrono::milliseconds>(endTimeCopy - startTimeCopy).count(),
        eCopyTime,
        std::chrono::duration_cast<std::chrono::nanoseconds>(endTimeCopy - startTimeCopy).count());
    printf("Compute time CUDA: %d s\n                   %d ms\n                   %f ms (CUDA events)\n                   %d ns\n",
        std::chrono::duration_cast<std::chrono::seconds>(endTimeCompute - startTimeCompute).count(),
        std::chrono::duration_cast<std::chrono::milliseconds>(endTimeCompute - startTimeCompute).count(),
        eComputeTime,
        std::chrono::duration_cast<std::chrono::nanoseconds>(endTimeCompute - startTimeCompute).count());
    printf("Recive time CUDA:  %d s\n                   %d ms\n                   %f ms (CUDA events)\n                   %d ns\n",
        std::chrono::duration_cast<std::chrono::seconds>(endTimeRecive - startTimeRecive).count(),
        std::chrono::duration_cast<std::chrono::milliseconds>(endTimeRecive - startTimeRecive).count(),
        eReciveTime,
        std::chrono::duration_cast<std::chrono::nanoseconds>(endTimeRecive - startTimeRecive).count());
    printf("Free time CUDA:    %d s\n                   %d ms\n                   %f ms (CUDA events)\n                   %d ns\n",
        std::chrono::duration_cast<std::chrono::seconds>(endTimeFree - startTimeFree).count(),
        std::chrono::duration_cast<std::chrono::milliseconds>(endTimeFree - startTimeFree).count(),
        eFreeTime,
        std::chrono::duration_cast<std::chrono::nanoseconds>(endTimeFree - startTimeFree).count());
    fflush(stdout);

    hipEventDestroy(eAllocStart);
    hipEventDestroy(eAllocStop);

    hipEventDestroy(eCopyStart);
    hipEventDestroy(eCopyStop);

    hipEventDestroy(eComputeStart);
    hipEventDestroy(eComputeStop);

    hipEventDestroy(eReciveStart);
    hipEventDestroy(eReciveStop);

    hipEventDestroy(eFreeStart);
    hipEventDestroy(eFreeStop);

    return cudaStatus;
}
